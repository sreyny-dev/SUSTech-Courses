#include <stdio.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#define TIME_START gettimeofday(&t_start, NULL);
#define TIME_END(name)    gettimeofday(&t_end, NULL); \
                    elapsedTime = (t_end.tv_sec - t_start.tv_sec) * 1000.0;   \
                    elapsedTime += (t_end.tv_usec - t_start.tv_usec) / 1000.0;  \
                    printf(#name " Time = %f ms.\n", elapsedTime);

typedef struct
{
    size_t rows;
    size_t cols;
    float * data; // CPU memory
    float * data_device; //GPU memory
} Matrix;

Matrix * createMatrix(size_t r, size_t c)
{
    size_t len = r * c;
    if(len == 0)
    {
        fprintf(stderr, "Invalid size. The input should be > 0.\n");
        return NULL;
    }
    Matrix * p  = (Matrix *) malloc(sizeof(Matrix));
    if (p == NULL)
    {
        fprintf(stderr, "Allocate host memory failed.\n");
        goto ERR_TAG;
    }
    p->rows = r;
    p->cols = c;
    p->data = (float*)malloc(sizeof(float)*len);
    if(p->data == NULL)
    {
        fprintf(stderr, "Allocate host memory failed.\n");
        goto ERR_TAG;
    }
    if (hipMalloc(&p->data_device, sizeof(float) * len) != hipSuccess)
    {
        fprintf(stderr, "Allocate device memory failed.\n");
        goto ERR_TAG;
    }
    return p;
    ERR_TAG:
    if(p && p->data) free(p->data);
    if(p) free(p);
    return NULL;
}

void freeMatrix(Matrix ** pp)
{
    if(pp == NULL) return;
    Matrix * p = *pp;
    if(p != NULL)
    {
        if(p->data) free(p->data);
        if(p->data_device) hipFree(p->data_device);
    }
    *pp = NULL;
}

// a simple function to set all elements to the same value
bool setMatrix(Matrix * pMat, float val)
{
    if(pMat == NULL)
    {
        fprintf(stderr, "NULL pointer.\n");
        return false;
    }
    size_t len = pMat->rows * pMat->cols;
    for(size_t i = 0; i < len; i++)
        pMat->data[i] = val;

    return true;
}

bool addCPU(const Matrix * pMat1, Matrix * pMat2, float a, float b)
{
    if( pMat1 == NULL || pMat2 == NULL)
    {
        fprintf(stderr, "Null pointer.\n");
        return false;
    }
    if (pMat1->rows != pMat2->rows || pMat1->cols != pMat2->cols)
    {
        fprintf(stderr, "The 2 matrices are not in the same size.\n");
        return false;
    }
    size_t len = pMat1->rows * pMat1->cols;
    for (int i = 0; i < len; i++)
        pMat2->data[i] = pMat1->data[i] * a + b;
    return true;
}

__global__ void addKernel(const float * input1, const float * input2, float * output, size_t len, float a, float b)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i < len)
        output[i] = input1[i] * a + b;
}

bool addGPU(const Matrix * pMat1, Matrix * pMat2, float a, float b)
{
    if( pMat1 == NULL || pMat2 == NULL)
    {
        fprintf(stderr, "Null pointer.\n");
        return false;
    }
    if (pMat1->rows != pMat2->rows || pMat1->cols != pMat2->cols)
    {
        fprintf(stderr, "The 2 matrices are not in the same size.\n");
        return false;
    }

    hipError_t ecode = hipSuccess;
    size_t len = pMat1->rows * pMat1->cols;

    hipMemcpy(pMat1->data_device, pMat1->data, sizeof(float)*len, hipMemcpyHostToDevice);
    addKernel<<<(len+255)/256, 256>>>(pMat1->data_device, pMat2->data_device, pMat2->data_device, len, a, b);
    if ((ecode = hipGetLastError()) != hipSuccess)
    {
        fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(ecode));
        return false;
    }
    hipMemcpy(pMat2->data, pMat2->data_device, sizeof(float)*len, hipMemcpyDeviceToHost);

    return true;
}

int main()
{
    struct timeval t_start, t_end;
    double elapsedTime = 0;

    int dev_count = 0;
    hipGetDeviceCount(&dev_count);
    if (dev_count == 0) {
        fprintf(stderr, "No CUDA devices found.\n");
        return -1;
    }

    int dev_id = 0; // Set to a valid device ID
    if (dev_id >= dev_count) {
        fprintf(stderr, "Invalid device ID %d. Only %d devices available.\n", dev_id, dev_count);
        return -1;
    }

    hipSetDevice(dev_id);
    hipGetDevice(&dev_id);
    printf("You have %d CUDA devices.\n", dev_count);
    printf("You are using device %d.\n", dev_id);

    Matrix * pMat1 = createMatrix(4096, 4096);
    Matrix * pMat2 = createMatrix(4096, 4096);
    Matrix * pMat3 = createMatrix(4096, 4096);

    if (pMat1 == NULL || pMat2 == NULL || pMat3 == NULL) {
        fprintf(stderr, "Matrix creation failed.\n");
        return -1;
    }

    setMatrix(pMat1, 1.0f);

    float a = 2.0f;
    float b = 3.0f;

    TIME_START
    addCPU(pMat1, pMat2, a, b);
    TIME_END(addCPU)
    printf("  Result = [%.1f, ..., %.1f]\n", pMat2->data[0], pMat2->data[pMat2->rows * pMat2->cols - 1]);

    TIME_START
    addGPU(pMat1, pMat2, a, b);
    TIME_END(addGPU)
    printf("  Result = [%.1f, ..., %.1f]\n", pMat2->data[0], pMat2->data[pMat2->rows * pMat2->cols - 1]);

    freeMatrix(&pMat1);
    freeMatrix(&pMat2);
    freeMatrix(&pMat3);

    return 0;
}
