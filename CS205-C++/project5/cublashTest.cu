#include <iostream>
#include <hip/hip_runtime.h>
#include <hipblas.h>

void checkCudaErrors(hipError_t result) {
    if (result != hipSuccess) {
        std::cerr << "CUDA Runtime Error: " << hipGetErrorString(result) << std::endl;
        exit(-1);
    }
}

void checkCublasErrors(hipblasStatus_t result) {
    if (result != HIPBLAS_STATUS_SUCCESS) {
        std::cerr << "cuBLAS Error: " << result << std::endl;
        exit(-1);
    }
}

int main() {
    const int N = 3;
    float h_A[N][N] = { {1.0f, 2.0f, 3.0f}, {4.0f, 5.0f, 6.0f}, {7.0f, 8.0f, 9.0f} };
    float h_B[N][N] = { {1.0f, 0.0f, 0.0f}, {0.0f, 1.0f, 0.0f}, {0.0f, 0.0f, 1.0f} };
    float h_C[N][N];

    float *d_A, *d_B, *d_C;
    hipblasHandle_t handle;

    checkCudaErrors(hipMalloc(&d_A, N * N * sizeof(float)));
    checkCudaErrors(hipMalloc(&d_B, N * N * sizeof(float)));
    checkCudaErrors(hipMalloc(&d_C, N * N * sizeof(float)));

    checkCudaErrors(hipMemcpy(d_A, h_A, N * N * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_B, h_B, N * N * sizeof(float), hipMemcpyHostToDevice));

    checkCublasErrors(hipblasCreate(&handle));
    const float alpha = 1.0f;
    const float beta = 0.0f;
    checkCublasErrors(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, d_A, N, d_B, N, &beta, d_C, N));

    checkCudaErrors(hipMemcpy(h_C, d_C, N * N * sizeof(float), hipMemcpyDeviceToHost));

    std::cout << "Result matrix C = A * B:" << std::endl;
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            std::cout << h_C[i][j] << " ";
        }
        std::cout << std::endl;
    }

    hipblasDestroy(handle);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
