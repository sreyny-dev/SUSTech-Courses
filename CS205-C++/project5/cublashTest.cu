#include <iostream>
#include <chrono>

#include <openblas/cblas.h>

#include <hip/hip_runtime.h>
#include <hipblas.h>

void matrixMultiplyOpenBLAS(int N) {
    float *A = new float[N * N];
    float *B = new float[N * N];
    float *C = new float[N * N];

    for (int i = 0; i < N * N; ++i) {
        A[i] = static_cast<float>(rand()) / RAND_MAX;
        B[i] = static_cast<float>(rand()) / RAND_MAX;
        C[i] = 0.0f;
    }

    auto start = std::chrono::high_resolution_clock::now();

    cblas_sgemm(CblasRowMajor, CblasNoTrans, CblasNoTrans, N, N, N, 1.0f, A, N, B, N, 0.0f, C, N);

    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float> duration = end - start;
    std::cout << "OpenBLAS: " << duration.count() << " seconds" << std::endl;

    delete[] A;
    delete[] B;
    delete[] C;
}

void matrixMultiplyCuBLAS(int N) {
    float *h_A = new float[N * N];
    float *h_B = new float[N * N];
    float *h_C = new float[N * N];

    for (int i = 0; i < N * N; ++i) {
        h_A[i] = static_cast<float>(rand()) / RAND_MAX;
        h_B[i] = static_cast<float>(rand()) / RAND_MAX;
        h_C[i] = 0.0f;
    }

    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, N * N * sizeof(float));
    hipMalloc((void**)&d_B, N * N * sizeof(float));
    hipMalloc((void**)&d_C, N * N * sizeof(float));

    hipMemcpy(d_A, h_A, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N * N * sizeof(float), hipMemcpyHostToDevice);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    auto start = std::chrono::high_resolution_clock::now();

    const float alpha = 1.0f;
    const float beta = 0.0f;
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, d_A, N, d_B, N, &beta, d_C, N);

    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float> duration = end - start;
    std::cout << "cuBLAS: " << duration.count() << " seconds" << std::endl;

    hipblasDestroy(handle);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
}

int main() {
    int N = 4096;
    std::cout << "Matrix multiplication with OpenBLAS and cuBLAS for " << N << "x" << N << " matrices\n";

    matrixMultiplyOpenBLAS(N);
    matrixMultiplyCuBLAS(N);

    return 0;
}
